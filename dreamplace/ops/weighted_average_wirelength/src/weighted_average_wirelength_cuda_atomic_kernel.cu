#include "hip/hip_runtime.h"
#include <cfloat>
#include <stdio.h>
#include "assert.h"
#include "hip/hip_runtime.h"
#include "print.h"
#include "functional_cuda.h"

template <typename T>
__global__ void computeExpSum(
        const T* exp_x, 
        const int* pin2net_map, 
        const unsigned char* net_mask, 
        int num_nets,
        int num_pins, 
        T* exp_x_sum
        )
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_pins; i += blockDim.x * gridDim.x) 
    {
        int net_id = pin2net_map[i]; 
        if (net_id >= 0 || net_mask[net_id])
        {
            atomicAdd(&exp_x_sum[net_id], exp_x[i]); 
            __syncthreads();
        }
    }
}

template <typename T>
__global__ void computeXExpSum(
        const T* x, 
        const T* exp_x, 
        const int* pin2net_map, 
        const unsigned char* net_mask, 
        int num_nets,
        int num_pins, 
        T* xexp_x_sum
        )
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_pins; i += blockDim.x * gridDim.x) 
    {
        int net_id = pin2net_map[i]; 
        if (net_id >= 0 || net_mask[net_id])
        {
            atomicAdd(&xexp_x_sum[net_id], x[i]*exp_x[i]); 
            __syncthreads();
        }
    }
}

template <typename T, typename V>
int computeWeightedAverageWirelengthCudaAtomicLauncher(
        const T* x, const T* y, 
        const int* pin2net_map, 
        const unsigned char* net_mask, 
        int num_nets,
        int num_pins, 
        const T* gamma, 
        T* exp_xy, T* exp_nxy, 
        T* exp_xy_sum, T* exp_nxy_sum, 
        T* xyexp_xy_sum, T* xyexp_nxy_sum, 
        V* xy_max, V* xy_min, 
        T* partial_wl, // wirelength of each net 
        const T* grad_tensor, 
        T* grad_x_tensor, T* grad_y_tensor // the gradient is partial total wirelength to partial pin position  
        )
{
    int thread_count = 1024; 
    int block_count = 32; // separate x and y

    hipError_t status; 
    hipStream_t stream_x_exp; 
    hipStream_t stream_nx_exp; 
    hipStream_t stream_y_exp; 
    hipStream_t stream_ny_exp; 
    status = hipStreamCreate(&stream_x_exp);
    if (status != hipSuccess)
    {
        printf("hipStreamCreate failed for stream_x_exp\n");
        fflush(stdout);
        return 1; 
    }
    status = hipStreamCreate(&stream_y_exp);
    if (status != hipSuccess)
    {
        printf("hipStreamCreate failed for stream_y_exp\n");
        fflush(stdout);
        return 1; 
    }

    if (grad_tensor)
    {
        computeWeightedAverageWirelengthGrad<<<block_count, thread_count, 0, stream_x_exp>>>(
                x, 
                exp_xy, exp_nxy, 
                exp_xy_sum, exp_nxy_sum, 
                xyexp_xy_sum, xyexp_nxy_sum, 
                pin2net_map, 
                net_mask, 
                num_nets,
                num_pins, 
                gamma, 
                grad_tensor, 
                grad_x_tensor
                );
        computeWeightedAverageWirelengthGrad<<<block_count, thread_count, 0, stream_y_exp>>>(
                y, 
                exp_xy+num_pins, exp_nxy+num_pins, 
                exp_xy_sum+num_nets, exp_nxy_sum+num_nets, 
                xyexp_xy_sum+num_nets, xyexp_nxy_sum+num_nets, 
                pin2net_map, 
                net_mask, 
                num_nets,
                num_pins, 
                gamma, 
                grad_tensor, 
                grad_y_tensor
                );
    }
    else
    {
        status = hipStreamCreate(&stream_nx_exp);
        if (status != hipSuccess)
        {
            printf("hipStreamCreate failed for stream_nx_exp\n");
            fflush(stdout);
            return 1; 
        }
        status = hipStreamCreate(&stream_ny_exp);
        if (status != hipSuccess)
        {
            printf("hipStreamCreate failed for stream_ny_exp\n");
            fflush(stdout);
            return 1; 
        }

        // compute max/min 
        computeMax<<<block_count, thread_count, 0, stream_x_exp>>>(
                x, 
                pin2net_map, 
                net_mask, 
                num_nets, 
                num_pins, 
                xy_max
                );
        computeMin<<<block_count, thread_count, 0, stream_nx_exp>>>(
                x, 
                pin2net_map, 
                net_mask, 
                num_nets, 
                num_pins, 
                xy_min
                );
        computeMax<<<block_count, thread_count, 0, stream_y_exp>>>(
                y, 
                pin2net_map, 
                net_mask, 
                num_nets, 
                num_pins, 
                xy_max+num_nets
                );
        computeMin<<<block_count, thread_count, 0, stream_ny_exp>>>(
                y, 
                pin2net_map, 
                net_mask, 
                num_nets, 
                num_pins, 
                xy_min+num_nets
                );

        // compute exp and negative exp 
        computeExp<<<block_count, thread_count, 0, stream_x_exp>>>(
                x, 
                pin2net_map, 
                net_mask, 
                num_nets,
                num_pins, 
                gamma, 
                xy_max, 
                exp_xy
                );
        computeNegExp<<<block_count, thread_count, 0, stream_nx_exp>>>(
                x, 
                pin2net_map, 
                net_mask, 
                num_nets,
                num_pins, 
                gamma, 
                xy_min, 
                exp_nxy
                );
        computeExp<<<block_count, thread_count, 0, stream_y_exp>>>(
                y, 
                pin2net_map, 
                net_mask, 
                num_nets,
                num_pins, 
                gamma, 
                xy_max+num_nets, 
                exp_xy+num_pins
                );
        computeNegExp<<<block_count, thread_count, 0, stream_ny_exp>>>(
                y, 
                pin2net_map, 
                net_mask, 
                num_nets,
                num_pins, 
                gamma, 
                xy_min+num_nets, 
                exp_nxy+num_pins
                );

        // compute exp sum 
        computeExpSum<<<block_count, thread_count, 0, stream_x_exp>>>(
                exp_xy, 
                pin2net_map, 
                net_mask, 
                num_nets,
                num_pins, 
                exp_xy_sum
                );
        computeExpSum<<<block_count, thread_count, 0, stream_nx_exp>>>(
                exp_nxy, 
                pin2net_map, 
                net_mask, 
                num_nets,
                num_pins, 
                exp_nxy_sum
                );
        computeExpSum<<<block_count, thread_count, 0, stream_y_exp>>>(
                exp_xy+num_pins, 
                pin2net_map, 
                net_mask, 
                num_nets,
                num_pins, 
                exp_xy_sum+num_nets
                );
        computeExpSum<<<block_count, thread_count, 0, stream_ny_exp>>>(
                exp_nxy+num_pins, 
                pin2net_map, 
                net_mask, 
                num_nets,
                num_pins, 
                exp_nxy_sum+num_nets
                );

        // compute x exp sum 
        computeXExpSum<<<block_count, thread_count, 0, stream_x_exp>>>(
                x, 
                exp_xy, 
                pin2net_map, 
                net_mask, 
                num_nets,
                num_pins, 
                xyexp_xy_sum
                );
        computeXExpSum<<<block_count, thread_count, 0, stream_nx_exp>>>(
                x, 
                exp_nxy, 
                pin2net_map, 
                net_mask, 
                num_nets,
                num_pins, 
                xyexp_nxy_sum
                );
        computeXExpSum<<<block_count, thread_count, 0, stream_y_exp>>>(
                y, 
                exp_xy+num_pins, 
                pin2net_map, 
                net_mask, 
                num_nets,
                num_pins, 
                xyexp_xy_sum+num_nets
                );
        computeXExpSum<<<block_count, thread_count, 0, stream_ny_exp>>>(
                y, 
                exp_nxy+num_pins, 
                pin2net_map, 
                net_mask, 
                num_nets,
                num_pins, 
                xyexp_nxy_sum+num_nets
                );

        // compute log sum exp 
        computeXExpSumByExpSum<<<block_count, thread_count, 0, stream_x_exp>>>(
                xyexp_xy_sum, 
                exp_xy_sum, 
                pin2net_map, 
                net_mask, 
                num_nets,
                gamma, 
                partial_wl
                );
        computeXNegExpSumByNegExpSum<<<block_count, thread_count, 0, stream_nx_exp>>>(
                xyexp_nxy_sum, 
                exp_nxy_sum, 
                pin2net_map, 
                net_mask, 
                num_nets,
                gamma, 
                partial_wl+num_nets
                );

        computeXExpSumByExpSum<<<block_count, thread_count, 0, stream_y_exp>>>(
                xyexp_xy_sum+num_nets, 
                exp_xy_sum+num_nets, 
                pin2net_map, 
                net_mask, 
                num_nets,
                gamma, 
                partial_wl+2*num_nets
                );
        computeXNegExpSumByNegExpSum<<<block_count, thread_count, 0, stream_ny_exp>>>(
                xyexp_nxy_sum+num_nets, 
                exp_nxy_sum+num_nets, 
                pin2net_map, 
                net_mask, 
                num_nets,
                gamma, 
                partial_wl+3*num_nets
                );

        // I move out the summation to use ATen 
        // significant speedup is observed 
        //sumArray<<<1, 1>>>(partial_wl, 2*num_nets, wl);

        status = hipStreamDestroy(stream_nx_exp); 
        stream_nx_exp = 0;
        if (status != hipSuccess) 
        {
            printf("stream_nx_exp destroy failed\n");
            fflush(stdout);
            return 1;
        }   
        status = hipStreamDestroy(stream_ny_exp); 
        stream_ny_exp = 0; 
        if (status != hipSuccess) 
        {
            printf("stream_ny_exp destroy failed\n");
            fflush(stdout);
            return 1;
        }   
    }

    /* destroy stream */
    status = hipStreamDestroy(stream_x_exp); 
    stream_x_exp = 0;
    if (status != hipSuccess) 
    {
        printf("stream_x_exp destroy failed\n");
        fflush(stdout);
        return 1;
    }   
    status = hipStreamDestroy(stream_y_exp); 
    stream_y_exp = 0; 
    if (status != hipSuccess) 
    {
        printf("stream_y_exp destroy failed\n");
        fflush(stdout);
        return 1;
    }   

    return 0; 
}


#define REGISTER_KERNEL_LAUNCHER(T, V) \
    int instantiateComputeWeightedAverageWirelengthAtomicLauncher(\
            const T* x, const T* y, \
            const int* pin2net_map, \
            const unsigned char* net_mask, \
            int num_nets, \
            int num_pins, \
            const T* gamma, \
            T* exp_xy, T* exp_nxy, \
            T* exp_xy_sum, T* exp_nxy_sum,\
            T* xyexp_xy_sum, T* xyexp_nxy_sum, \
            V* xy_max, V* xy_min, \
            T* partial_wl, \
            const T* grad_tensor, \
            T* grad_x_tensor, T* grad_y_tensor \
            )\
    {\
        return computeWeightedAverageWirelengthCudaAtomicLauncher(\
                x, y, \
                pin2net_map, \
                net_mask, \
                num_nets,\
                num_pins,\
                gamma, \
                exp_xy, exp_nxy, \
                exp_xy_sum, exp_nxy_sum, \
                xyexp_xy_sum, xyexp_nxy_sum, \
                xy_max, xy_min, \
                partial_wl, \
                grad_tensor, \
                grad_x_tensor, grad_y_tensor  \
                );\
    }
REGISTER_KERNEL_LAUNCHER(float, int);
REGISTER_KERNEL_LAUNCHER(double, int);
