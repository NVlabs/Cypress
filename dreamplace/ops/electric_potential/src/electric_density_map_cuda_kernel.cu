#include "hip/hip_runtime.h"
/**
 * @file   electric_density_map_cuda_kernel.cu
 * @author Yibo Lin
 * @date   Aug 2018
 */
#include <stdio.h>
#include <math.h>
#include <float.h>
#include "hip/hip_runtime.h"
#include "utility/src/print.h"
#include "utility/src/Msg.h"
#include "utility/src/utils.cuh"
#include "electric_potential/src/density_function.h"

DREAMPLACE_BEGIN_NAMESPACE

/// define triangle_density_function
template <typename T> 
inline __device__ 
DEFINE_TRIANGLE_DENSITY_FUNCTION(T);
/// define exact_density_function
template <typename T> 
inline __device__ 
DEFINE_EXACT_DENSITY_FUNCTION(T);

template <typename T>
__global__ void __launch_bounds__(1024, 8) computeTriangleDensityMap(
    const T *x_tensor, const T *y_tensor,
    const T *node_size_x_clamped_tensor, const T *node_size_y_clamped_tensor,
    const T *offset_x_tensor, const T *offset_y_tensor,
    const T *ratio_tensor,
    const T *bin_center_x_tensor, const T *bin_center_y_tensor,
    const int num_nodes,
    const int num_bins_x, const int num_bins_y,
    const T xl, const T yl, const T xh, const T yh,
    const T half_bin_size_x, const T half_bin_size_y,
    const T bin_size_x, const T bin_size_y,
    const T inv_bin_size_x, const T inv_bin_size_y,
    T *density_map_tensor,
    const int *sorted_node_map ///< can be NULL if not sorted 
    )
{
    int index = blockIdx.x * blockDim.z + threadIdx.z;
    if (index < num_nodes)
    {
        int i = (sorted_node_map)? sorted_node_map[index] : index;

        // use stretched node size 
        T node_size_x = node_size_x_clamped_tensor[i];
        T node_size_y = node_size_y_clamped_tensor[i];
        T node_x = x_tensor[i] + offset_x_tensor[i];
        T node_y = y_tensor[i] + offset_y_tensor[i];
        T ratio = ratio_tensor[i];

        int bin_index_xl = int((node_x - xl) * inv_bin_size_x);
        int bin_index_xh = int(((node_x + node_size_x - xl) * inv_bin_size_x)) + 1; // exclusive
        bin_index_xl = DREAMPLACE_STD_NAMESPACE::max(bin_index_xl, 0);
        bin_index_xh = DREAMPLACE_STD_NAMESPACE::min(bin_index_xh, num_bins_x);

        int bin_index_yl = int((node_y - yl) * inv_bin_size_y);
        int bin_index_yh = int(((node_y + node_size_y - yl) * inv_bin_size_y)) + 1; // exclusive
        bin_index_yl = DREAMPLACE_STD_NAMESPACE::max(bin_index_yl, 0);
        bin_index_yh = DREAMPLACE_STD_NAMESPACE::min(bin_index_yh, num_bins_y);

        // update density potential map
        for (int k = bin_index_xl + threadIdx.y; k < bin_index_xh; k += blockDim.y)
        {
            T px = triangle_density_function(node_x, node_size_x, xl, k, bin_size_x);
            T px_by_ratio = px * ratio;

            for (int h = bin_index_yl + threadIdx.x; h < bin_index_yh; h += blockDim.x)
            {
                T py = triangle_density_function(node_y, node_size_y, yl, h, bin_size_y);
                T area = px_by_ratio * py;
                atomicAdd(&density_map_tensor[k * num_bins_y + h], area);
            }
        }
    }
}

/// @brief An unrolled way to compute the density map. 
/// Currently it is not as efficient as computeTriangleDensityMap, 
/// it has the potential to be better. 
/// It is not used for now. 
template <typename T>
__global__ void computeTriangleDensityMapUnroll(
    const T *x_tensor, const T *y_tensor,
    const T *node_size_x_clamped_tensor, const T *node_size_y_clamped_tensor,
    const T *offset_x_tensor, const T *offset_y_tensor,
    const T *ratio_tensor,
    const T *bin_center_x_tensor, const T *bin_center_y_tensor,
    const int num_nodes,
    const int num_bins_x, const int num_bins_y,
    const T xl, const T yl, const T xh, const T yh,
    const T half_bin_size_x, const T half_bin_size_y,
    const T bin_size_x, const T bin_size_y,
    const T inv_bin_size_x, const T inv_bin_size_y,
    T *density_map_tensor,
    const int *sorted_node_map ///< can be NULL if not sorted 
    )
{
    int index = blockIdx.x * blockDim.y + threadIdx.y;
    if (index < num_nodes)
    {
        int i = (sorted_node_map)? sorted_node_map[index] : index;

        T node_size_x = node_size_x_clamped_tensor[i];
        T node_size_y = node_size_y_clamped_tensor[i];
        T node_x = x_tensor[i] + offset_x_tensor[i];
        T node_y = y_tensor[i] + offset_y_tensor[i];
        T ratio = ratio_tensor[i];

        int bin_index_xl = int((node_x - xl) * inv_bin_size_x);
        int bin_index_xh = int(((node_x + node_size_x - xl) * inv_bin_size_x)); // inclusive
        bin_index_xl = DREAMPLACE_STD_NAMESPACE::max(bin_index_xl, 0);
        bin_index_xh = DREAMPLACE_STD_NAMESPACE::min(bin_index_xh, num_bins_x - 1);

        int bin_index_yl = int((node_y - yl) * inv_bin_size_y);
        int bin_index_yh = int(((node_y + node_size_y - yl) * inv_bin_size_y)); // inclusive
        bin_index_yl = DREAMPLACE_STD_NAMESPACE::max(bin_index_yl, 0);
        bin_index_yh = DREAMPLACE_STD_NAMESPACE::min(bin_index_yh, num_bins_y - 1);

        // update density potential map
        int k, h;

        int cond = ((bin_index_xl == bin_index_xh) << 1) | (bin_index_yl == bin_index_yh);
        switch (cond)
        {
        case 0:
        {
            T px_c = bin_size_x;

            T py_l = yl + bin_index_yl * bin_size_y + bin_size_y - node_y;
            T py_c = bin_size_y;
            T py_h = node_y + node_size_y - (bin_index_yh * bin_size_y + yl);

            T area_xc_yl = px_c * py_l * ratio;
            T area_xc_yc = px_c * py_c * ratio;
            T area_xc_yh = px_c * py_h * ratio;

            k = bin_index_xl;

            if (threadIdx.x == 0)
            {
                T px_l = xl + bin_index_xl * bin_size_x + bin_size_x - node_x;
                T area_xl_yl = px_l * py_l * ratio;
                T area_xl_yc = px_l * py_c * ratio;
                T area_xl_yh = px_l * py_h * ratio;
                h = bin_index_yl;
                atomicAdd(&density_map_tensor[k * num_bins_y + h], area_xl_yl);
                for (++h; h < bin_index_yh; ++h)
                {
                    atomicAdd(&density_map_tensor[k * num_bins_y + h], area_xl_yc);
                }
                atomicAdd(&density_map_tensor[k * num_bins_y + h], area_xl_yh);
                k += blockDim.x;
            }

            for (k += threadIdx.x; k < bin_index_xh; k += blockDim.x)
            {
                h = bin_index_yl;
                atomicAdd(&density_map_tensor[k * num_bins_y + h], area_xc_yl);
                for (++h; h < bin_index_yh; ++h)
                {
                    atomicAdd(&density_map_tensor[k * num_bins_y + h], area_xc_yc);
                }
                atomicAdd(&density_map_tensor[k * num_bins_y + h], area_xc_yh);
            }

            if (k == bin_index_xh)
            {
                T px_h = node_x + node_size_x - (bin_index_xh * bin_size_x + xl);
                T area_xh_yl = px_h * py_l * ratio;
                T area_xh_yc = px_h * py_c * ratio;
                T area_xh_yh = px_h * py_h * ratio;
                h = bin_index_yl;
                atomicAdd(&density_map_tensor[k * num_bins_y + h], area_xh_yl);
                for (++h; h < bin_index_yh; ++h)
                {
                    atomicAdd(&density_map_tensor[k * num_bins_y + h], area_xh_yc);
                }
                atomicAdd(&density_map_tensor[k * num_bins_y + h], area_xh_yh);
            }

            return;
        }
        case 1:
        {
            T py = yl + bin_index_yl * bin_size_y + bin_size_y - node_y;
            h = bin_index_yl;
            k = bin_index_xl;

            if (threadIdx.x == 0)
            {
                T px_l = xl + bin_index_xl * bin_size_x + bin_size_x - node_x;
                T area_xl = px_l * py * ratio;
                atomicAdd(&density_map_tensor[k * num_bins_y + h], area_xl);
                k += blockDim.x;
            }

            T px_c = bin_size_x;
            T area_xc = px_c * py * ratio;
            for (k += threadIdx.x; k < bin_index_xh; k += blockDim.x)
            {
                atomicAdd(&density_map_tensor[k * num_bins_y + h], area_xc);
            }

            if (k == bin_index_xh)
            {
                T px_h = node_x + node_size_x - (bin_index_xh * bin_size_x + xl);
                T area_xh = px_h * py * ratio;
                atomicAdd(&density_map_tensor[k * num_bins_y + h], area_xh);
            }

            return;
        }
        case 2:
        {
            T px = xl + bin_index_xl * bin_size_x + bin_size_x - node_x;
            k = bin_index_xl;
            h = bin_index_yl;

            if (threadIdx.x == 0)
            {
                T py_l = yl + bin_index_yl * bin_size_y + bin_size_y - node_y;
                T area_yl = px * py_l * ratio;
                atomicAdd(&density_map_tensor[k * num_bins_y + h], area_yl);
                h += blockDim.x;
            }

            T py_c = bin_size_y;
            T area_yc = px * py_c * ratio;
            for (h += threadIdx.x; h < bin_index_yh; h += blockDim.x)
            {
                atomicAdd(&density_map_tensor[k * num_bins_y + h], area_yc);
            }

            if (h == bin_index_yh)
            {
                T py_h = node_y + node_size_y - (bin_index_yh * bin_size_y + yl);
                T area_yh = px * py_h * ratio;
                atomicAdd(&density_map_tensor[k * num_bins_y + h], area_yh);
            }

            return;
        }
        case 3:
        {
            if (threadIdx.x == 0)
            {
                T px = xl + bin_index_xl * bin_size_x + bin_size_x - node_x;
                T py = yl + bin_index_yl * bin_size_y + bin_size_y - node_y;
                T area = px * py * ratio;

                k = bin_index_xl;
                h = bin_index_yl;
                atomicAdd(&density_map_tensor[k * num_bins_y + h], area);
            }
            return;
        }
        default:
            assert(0);
        }
    }
}

/// @brief Compute exact density map using cell-to-bin parallelization strategy
template <typename T>
__global__ void computeExactDensityMap(
        const T* flat_fixed_node_boxes, 
        const T* bin_center_x_tensor, const T* bin_center_y_tensor,
        const int num_boxes,
        const int num_bins_x, const int num_bins_y,
        const T xl, const T yl, const T xh, const T yh,
        const T bin_size_x, const T bin_size_y,
        const int num_impacted_bins_x, const int num_impacted_bins_y,
        bool fixed_node_flag,
        T *density_map_tensor
    )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // rank-one update density map
    if (i < num_boxes * num_impacted_bins_x * num_impacted_bins_y)
    {
        int box_id = i / (num_impacted_bins_x * num_impacted_bins_y);
        int residual_index = i - box_id * num_impacted_bins_x * num_impacted_bins_y;
        auto box = flat_fixed_node_boxes + i * 4; 
        T bxl = box[0]; 
        T byl = box[1];
        T bxh = box[2]; 
        T byh = box[3];
        // x direction
        int bin_index_xl = int((bxl - xl) / bin_size_x);
        bin_index_xl = DREAMPLACE_STD_NAMESPACE::max(bin_index_xl, 0);
        int k = bin_index_xl + int(residual_index / num_impacted_bins_y);
        if (k + 1 > num_bins_x)
        {
            return;
        }
        // y direction
        int bin_index_yl = int((byl - yl) / bin_size_y);
        bin_index_yl = DREAMPLACE_STD_NAMESPACE::max(bin_index_yl, 0);
        int h = bin_index_yl + (residual_index % num_impacted_bins_y);
        if (h + 1 > num_bins_y)
        {
            return;
        }

        T px = exact_density_function(bxl, bxh - bxl, bin_center_x_tensor[k], bin_size_x, xl, xh, fixed_node_flag);
        T py = exact_density_function(byl, byh - byl, bin_center_y_tensor[h], bin_size_y, yl, yh, fixed_node_flag);

        // still area 
        atomicAdd(&density_map_tensor[k*num_bins_y+h], px*py); 
    }
}

/// @brief Compute exact density map using cell-by-cell parallelization strategy
template <typename T>
__global__ void computeExactDensityMapCellByCell(
        const T* flat_fixed_node_boxes, 
        const T* bin_center_x_tensor, const T* bin_center_y_tensor,
        const int num_boxes,
        const int num_bins_x, const int num_bins_y,
        const T xl, const T yl, const T xh, const T yh,
        const T bin_size_x, const T bin_size_y,
        bool fixed_node_flag,
        T* density_map_tensor
        )
{
    auto box2bin = [&](T bxl, T byl, T bxh, T byh, T* buf_map){
        // x direction
        int bin_index_xl = int((bxl-xl)/bin_size_x);
        int bin_index_xh = int(ceil((bxh-xl)/bin_size_x))+1; // exclusive
        bin_index_xl = DREAMPLACE_STD_NAMESPACE::max(bin_index_xl, 0);
        bin_index_xh = DREAMPLACE_STD_NAMESPACE::min(bin_index_xh, num_bins_x);

        // y direction
        int bin_index_yl = int((byl-yl)/bin_size_y);
        int bin_index_yh = int(ceil((byh-yl)/bin_size_y))+1; // exclusive
        bin_index_yl = DREAMPLACE_STD_NAMESPACE::max(bin_index_yl, 0);
        bin_index_yh = DREAMPLACE_STD_NAMESPACE::min(bin_index_yh, num_bins_y);

        for (int k = bin_index_xl; k < bin_index_xh; ++k)
        {
            T px = exact_density_function(bxl, bxh-bxl, bin_center_x_tensor[k], bin_size_x, xl, xh, fixed_node_flag);
            for (int h = bin_index_yl; h < bin_index_yh; ++h)
            {
                T py = exact_density_function(byl, byh-byl, bin_center_y_tensor[h], bin_size_y, yl, yh, fixed_node_flag);

                // still area
                atomicAdd(&buf_map[k*num_bins_y+h], px * py);
            }
        }
    };

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_boxes)
    {
        auto box = flat_fixed_node_boxes + i * 4; 
        box2bin(box[0], box[1], box[2], box[3], density_map_tensor);
    }
}

template <typename T>
int computeTriangleDensityMapCudaLauncher(
    const T *x_tensor, const T *y_tensor,
    const T *node_size_x_clamped_tensor, const T *node_size_y_clamped_tensor,
    const T *offset_x_tensor, const T *offset_y_tensor,
    const T *ratio_tensor,
    const T *bin_center_x_tensor, const T *bin_center_y_tensor,
    int num_nodes, 
    const int num_bins_x, const int num_bins_y,
    int num_impacted_bins_x, int num_impacted_bins_y,
    const T xl, const T yl, const T xh, const T yh,
    const T bin_size_x, const T bin_size_y,
    T *density_map_tensor,
    const int *sorted_node_map)
{
    int thread_count = 64;
    // dim3 blockSize(4, thread_count, 1);
    dim3 blockSize(2, 2, thread_count);

    int block_count = (num_nodes - 1 + thread_count) / thread_count;
    computeTriangleDensityMap<<<block_count, blockSize>>>(
        x_tensor, y_tensor,
        node_size_x_clamped_tensor, node_size_y_clamped_tensor,
        offset_x_tensor, offset_y_tensor,
        ratio_tensor,
        bin_center_x_tensor, bin_center_y_tensor,
        num_nodes,
        num_bins_x, num_bins_y,
        xl, yl, xh, yh,
        bin_size_x / 2, bin_size_y / 2,
        bin_size_x, bin_size_y,
        1 / bin_size_x, 1 / bin_size_y,
        density_map_tensor,
        sorted_node_map
        );

    return 0;
}

/// @brief Compute exact density map using cell-to-bin parallelization strategy
template <typename T>
int computeExactDensityMapCudaLauncher(
        const T* flat_fixed_node_boxes, 
        const T* bin_center_x_tensor, const T* bin_center_y_tensor,
        const int num_boxes,
        const int num_bins_x, const int num_bins_y,
        const int num_impacted_bins_x, const int num_impacted_bins_y,
        const T xl, const T yl, const T xh, const T yh,
        const T bin_size_x, const T bin_size_y,
        bool fixed_node_flag,
        T *density_map_tensor
    )
{
    int thread_count = 512;
    int block_count = (num_boxes * num_impacted_bins_x * num_impacted_bins_y - 1 + thread_count) / thread_count;

    computeExactDensityMap<<<block_count, thread_count>>>(
            flat_fixed_node_boxes, 
            bin_center_x_tensor, bin_center_y_tensor,
            num_boxes,
            num_bins_x, num_bins_y,
            xl, yl, xh, yh,
            bin_size_x, bin_size_y,
            num_impacted_bins_x, num_impacted_bins_y,
            fixed_node_flag,
            density_map_tensor
        );

    return 0;
}

/// @brief Compute exact density map using cell-by-cell parallelization strategy
template <typename T>
int computeExactDensityMapCellByCellCudaLauncher(
        const T* flat_fixed_node_boxes, 
        const T* bin_center_x_tensor, const T* bin_center_y_tensor,
        const int num_boxes,
        const int num_bins_x, const int num_bins_y,
        const int num_impacted_bins_x, const int num_impacted_bins_y,
        const T xl, const T yl, const T xh, const T yh,
        const T bin_size_x, const T bin_size_y,
        bool fixed_node_flag,
        T *density_map_tensor
        )
{
    int thread_count = 256;
    int block_count = (num_boxes - 1 + thread_count) / thread_count;

    computeExactDensityMapCellByCell<<<block_count, thread_count>>>(
            flat_fixed_node_boxes, 
            bin_center_x_tensor, bin_center_y_tensor,
            num_boxes,
            num_bins_x, num_bins_y,
            xl, yl, xh, yh,
            bin_size_x, bin_size_y,
            fixed_node_flag,
            density_map_tensor
        );

    return 0;
}

#define REGISTER_KERNEL_LAUNCHER(T)                                                   \
    int instantiateComputeTriangleDensityMapLauncher(                                 \
        const T *x_tensor, const T *y_tensor,                                         \
        const T *node_size_x_clamped_tensor, const T *node_size_y_clamped_tensor,     \
        const T *offset_x_tensor, const T *offset_y_tensor,                           \
        const T *ratio_tensor,                                                        \
        const T *bin_center_x_tensor, const T *bin_center_y_tensor,                   \
        const int num_nodes,                                                          \
        const int num_bins_x, const int num_bins_y,                                   \
        const int num_impacted_bins_x, const int num_impacted_bins_y,                 \
        const T xl, const T yl, const T xh, const T yh,                               \
        const T bin_size_x, const T bin_size_y,                                       \
        T *density_map_tensor,                                                        \
        const int *sorted_node_map)                                                   \
    {                                                                                 \
        return computeTriangleDensityMapCudaLauncher(                                 \
            x_tensor, y_tensor,                                                       \
            node_size_x_clamped_tensor, node_size_y_clamped_tensor,                   \
            offset_x_tensor, offset_y_tensor,                                         \
            ratio_tensor,                                                             \
            bin_center_x_tensor, bin_center_y_tensor,                                 \
            num_nodes,                                                                \
            num_bins_x, num_bins_y,                                                   \
            num_impacted_bins_x, num_impacted_bins_y,                                 \
            xl, yl, xh, yh,                                                           \
            bin_size_x, bin_size_y,                                                   \
            density_map_tensor,                                                       \
            sorted_node_map);                                                         \
    }                                                                                 \
                                                                                      \
    int instantiateComputeExactDensityMapLauncher(                                    \
        const T* flat_fixed_node_boxes,                                               \
        const T *bin_center_x_tensor, const T *bin_center_y_tensor,                   \
        const int num_boxes,                                                          \
        const int num_bins_x, const int num_bins_y,                                   \
        const int num_impacted_bins_x, const int num_impacted_bins_y,                 \
        const T xl, const T yl, const T xh, const T yh,                               \
        const T bin_size_x, const T bin_size_y,                                       \
        bool fixed_node_flag,                                                         \
        T *density_map_tensor)                                                        \
    {                                                                                 \
        int a = computeExactDensityMapCudaLauncher(                                   \
            flat_fixed_node_boxes,                                                    \
            bin_center_x_tensor, bin_center_y_tensor,                                 \
            num_boxes,                                                                \
            num_bins_x, num_bins_y,                                                   \
            num_impacted_bins_x, num_impacted_bins_y,                                 \
            xl, yl, xh, yh,                                                           \
            bin_size_x, bin_size_y,                                                   \
            fixed_node_flag,                                                          \
            density_map_tensor);                                                      \
        int b = computeExactDensityMapCellByCellCudaLauncher(                         \
            flat_fixed_node_boxes,                                                    \
            bin_center_x_tensor, bin_center_y_tensor,                                 \
            num_boxes,                                                                \
            num_bins_x, num_bins_y,                                                   \
            num_impacted_bins_x, num_impacted_bins_y,                                 \
            xl, yl, xh, yh,                                                           \
            bin_size_x, bin_size_y,                                                   \
            fixed_node_flag,                                                          \
            density_map_tensor);                                                      \
        return a && b;                                                                \
    }

REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
